#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

#define CUDA_CHECK(x) if((x)!=hipSuccess){ \
    std::cerr<<"CUDA error at "<<__LINE__<<": "<<hipGetErrorString(x)<<std::endl; return -1;}
#define CUSOLVER_CHECK(x) if((x)!=HIPSOLVER_STATUS_SUCCESS){ \
    std::cerr<<"cuSOLVER error at "<<__LINE__<<std::endl; return -1;}

int main() {
    int m=3, n=2;
    std::vector<float> h_A = {1, 0,
                              0, 1,
                              1, 1}; // row-major m x n

    float *d_A;
    CUDA_CHECK(hipMalloc(&d_A, m*n*sizeof(float)));
    CUDA_CHECK(hipMemcpy(d_A, h_A.data(), m*n*sizeof(float), hipMemcpyHostToDevice));

    hipsolverHandle_t solverH;
    CUSOLVER_CHECK(hipsolverDnCreate(&solverH));

    int lwork=0;
    CUSOLVER_CHECK(hipsolverDnSgesvd_bufferSize(solverH, m, n, &lwork));

    float *d_S, *d_U, *d_VT, *d_work; int *devInfo;
    CUDA_CHECK(hipMalloc(&d_S, n*sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_U, m*m*sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_VT, n*n*sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_work, lwork*sizeof(float)));
    CUDA_CHECK(hipMalloc(&devInfo, sizeof(int)));

    signed char jobu = 'A', jobvt = 'A';
    CUSOLVER_CHECK(hipsolverDnSgesvd(solverH, jobu, jobvt, m, n, d_A, m, d_S, d_U, m, d_VT, n, d_work, lwork, nullptr, devInfo));

    std::vector<float> h_S(n);
    CUDA_CHECK(hipMemcpy(h_S.data(), d_S, n*sizeof(float), hipMemcpyDeviceToHost));

    std::cout << "Singular values:\n";
    for(auto s:h_S) std::cout << s << " ";
    std::cout << std::endl;

    hipFree(d_A); hipFree(d_S); hipFree(d_U); hipFree(d_VT); hipFree(d_work); hipFree(devInfo);
    hipsolverDnDestroy(solverH);

    return 0;
}

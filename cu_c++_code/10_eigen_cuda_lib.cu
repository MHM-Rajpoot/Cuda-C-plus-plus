#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <cmath>

#define CUDA_CHECK(x) if((x)!=hipSuccess){ \
    std::cerr<<"CUDA error at "<<__LINE__<<": "<<hipGetErrorString(x)<<std::endl; return -1;}
#define CUSOLVER_CHECK(x) if((x)!=HIPSOLVER_STATUS_SUCCESS){ \
    std::cerr<<"cuSOLVER error at "<<__LINE__<<std::endl; return -1;}

int main() {
    // Symmetric matrix 3x3
    std::vector<float> h_A = {4, 1, 1,
                              1, 3, 0,
                              1, 0, 2}; // row-major
    int n = 3;

    // Device memory
    float *d_A;
    CUDA_CHECK(hipMalloc(&d_A, n*n*sizeof(float)));
    CUDA_CHECK(hipMemcpy(d_A, h_A.data(), n*n*sizeof(float), hipMemcpyHostToDevice));

    // cuSOLVER handle
    hipsolverHandle_t solverH;
    CUSOLVER_CHECK(hipsolverDnCreate(&solverH));

    // Workspace
    int lwork = 0;
    CUSOLVER_CHECK(hipsolverDnSsyevd_bufferSize(solverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, n, d_A, n, nullptr, &lwork));

    float *d_W, *d_work;
    int *devInfo;
    CUDA_CHECK(hipMalloc(&d_W, n*sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_work, lwork*sizeof(float)));
    CUDA_CHECK(hipMalloc(&devInfo, sizeof(int)));

    // Compute eigenvalues and eigenvectors
    CUSOLVER_CHECK(hipsolverDnSsyevd(solverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, n, d_A, n, d_W, d_work, lwork, devInfo));

    int h_info;
    CUDA_CHECK(hipMemcpy(&h_info, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    if (h_info != 0) {
        std::cerr << "Eigen decomposition failed, info = " << h_info << std::endl;
        return -1;
    }

    // Copy results to host
    std::vector<float> h_W(n);
    std::vector<float> h_V(n*n);
    CUDA_CHECK(hipMemcpy(h_W.data(), d_W, n*sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_V.data(), d_A, n*n*sizeof(float), hipMemcpyDeviceToHost));

    std::cout << "Eigenvalues:\n";
    for(auto w:h_W) std::cout << w << " ";
    std::cout << "\nEigenvectors (columns):\n";
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++) std::cout << h_V[j + i*n] << " ";
        std::cout << "\n";
    }

    // Cleanup
    hipFree(d_A); hipFree(d_W); hipFree(d_work); hipFree(devInfo);
    hipsolverDnDestroy(solverH);

    return 0;
}

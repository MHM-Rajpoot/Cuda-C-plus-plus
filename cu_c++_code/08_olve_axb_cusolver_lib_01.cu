#include <iostream>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

#define CUDA_CHECK(x) if((x)!=hipSuccess){ \
    std::cerr<<"CUDA error at "<<__LINE__<<": "<<hipGetErrorString(x)<<std::endl; return -1;}
#define CUSOLVER_CHECK(x) if((x)!=HIPSOLVER_STATUS_SUCCESS){ \
    std::cerr<<"cuSOLVER error at "<<__LINE__<<std::endl; return -1;}

int main() {
    int n = 3; // matrix size

    // Example matrix A (row-major)
    std::vector<float> h_A = {4, 1, 0,
                              1, 3, 1,
                              0, 1, 2};
    // Right-hand side b
    std::vector<float> h_b = {1, 2, 3};

    // Device memory
    float *d_A = nullptr, *d_b = nullptr;
    int *d_ipiv = nullptr, *d_info = nullptr;
    CUDA_CHECK(hipMalloc(&d_A, n*n*sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_b, n*sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_ipiv, n*sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_info, sizeof(int)));

    // Copy A and b to device
    CUDA_CHECK(hipMemcpy(d_A, h_A.data(), n*n*sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, h_b.data(), n*sizeof(float), hipMemcpyHostToDevice));

    // cuSOLVER handle
    hipsolverHandle_t cusolverH;
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));

    // Workspace query
    int work_size = 0;
    CUSOLVER_CHECK(hipsolverDnSgetrf_bufferSize(cusolverH, n, n, d_A, n, &work_size));

    float *d_work = nullptr;
    CUDA_CHECK(hipMalloc(&d_work, work_size * sizeof(float)));

    // --- LU factorization ---
    CUSOLVER_CHECK(hipsolverDnSgetrf(cusolverH, n, n, d_A, n, d_work, d_ipiv, d_info));

    int h_info = 0;
    CUDA_CHECK(hipMemcpy(&h_info, d_info, sizeof(int), hipMemcpyDeviceToHost));
    if (h_info != 0) {
        std::cerr << "LU factorization failed, info = " << h_info << std::endl;
        return -1;
    }

    // --- Solve Ax = b ---
    CUSOLVER_CHECK(hipsolverDnSgetrs(cusolverH, HIPBLAS_OP_N, n, 1, d_A, n, d_ipiv, d_b, n, d_info));
    CUDA_CHECK(hipMemcpy(&h_info, d_info, sizeof(int), hipMemcpyDeviceToHost));
    if (h_info != 0) {
        std::cerr << "Solve failed, info = " << h_info << std::endl;
        return -1;
    }

    // Copy solution back to host
    std::vector<float> h_x(n);
    CUDA_CHECK(hipMemcpy(h_x.data(), d_b, n*sizeof(float), hipMemcpyDeviceToHost));

    std::cout << "Solution x = [ ";
    for (auto v : h_x) std::cout << v << " ";
    std::cout << "]\n";

    // --- Compute residual norm ||b - Ax|| in double precision ---
    std::vector<double> h_xd(n), h_Ad(n*n), h_bd(n);
    for (int i=0;i<n;i++) h_xd[i] = static_cast<double>(h_x[i]);
    for (int i=0;i<n*n;i++) h_Ad[i] = static_cast<double>(h_A[i]);
    for (int i=0;i<n;i++) h_bd[i] = static_cast<double>(h_b[i]);

    std::vector<double> res(n,0.0);
    for (int i=0;i<n;i++) {
        double sum=0.0;
        for (int j=0;j<n;j++) sum += h_Ad[i*n + j]*h_xd[j];
        res[i] = h_bd[i] - sum;
    }
    double rnorm=0.0;
    for (int i=0;i<n;i++) rnorm += res[i]*res[i];
    rnorm = std::sqrt(rnorm);

    std::cout << "Residual norm ||b - Ax|| = " << rnorm << std::endl;

    // Cleanup
    hipFree(d_A); hipFree(d_b); hipFree(d_ipiv); hipFree(d_info); hipFree(d_work);
    hipsolverDnDestroy(cusolverH);

    return 0;
}

#include <iostream>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define CUDA_CHECK(x) if((x)!=hipSuccess){ \
    std::cerr<<"CUDA error at "<<__LINE__<<": "<<hipGetErrorString(x)<<std::endl; return -1;}
#define CUBLAS_CHECK(x) if((x)!=HIPBLAS_STATUS_SUCCESS){ \
    std::cerr<<"cuBLAS error at "<<__LINE__<<std::endl; return -1;}

// Gram-Schmidt orthonormalization on host
void gram_schmidt(std::vector<float>& v1, std::vector<float>& v2) {
    // Normalize v1
    float norm1 = std::sqrt(v1[0]*v1[0]+v1[1]*v1[1]+v1[2]*v1[2]);
    for(int i=0;i<3;i++) v1[i] /= norm1;

    // v2 = v2 - proj_v1(v2)
    float dot = v1[0]*v2[0] + v1[1]*v2[1] + v1[2]*v2[2];
    for(int i=0;i<3;i++) v2[i] -= dot*v1[i];

    // Normalize v2
    float norm2 = std::sqrt(v2[0]*v2[0]+v2[1]*v2[1]+v2[2]*v2[2]);
    for(int i=0;i<3;i++) v2[i] /= norm2;
}

int main() {
    // 3D vectors spanning subspace
    std::vector<float> v1 = {1,1,0};
    std::vector<float> v2 = {1,0,1};
    gram_schmidt(v1,v2);

    // Vector to project
    std::vector<float> b = {3,2,1};

    // Device memory
    float *d_Q, *d_b, *d_proj;
    CUDA_CHECK(hipMalloc(&d_Q, 3*2*sizeof(float))); // 2 orthonormal vectors
    CUDA_CHECK(hipMalloc(&d_b, 3*sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_proj, 3*sizeof(float)));

    // Copy Q and b
    std::vector<float> h_Q = {v1[0], v1[1], v1[2],
                              v2[0], v2[1], v2[2]}; // column-major
    CUDA_CHECK(hipMemcpy(d_Q, h_Q.data(), 3*2*sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, b.data(), 3*sizeof(float), hipMemcpyHostToDevice));

    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    float alpha=1.0f, beta=0.0f;
    // proj = Q * (Q^T * b)
    float *d_temp;
    CUDA_CHECK(hipMalloc(&d_temp, 2*sizeof(float))); // 2x1 vector
    // d_temp = Q^T * b
    CUBLAS_CHECK(hipblasSgemv(handle, HIPBLAS_OP_T, 3, 2, &alpha, d_Q, 3, d_b, 1, &beta, d_temp, 1));
    // d_proj = Q * d_temp
    CUBLAS_CHECK(hipblasSgemv(handle, HIPBLAS_OP_N, 3, 2, &alpha, d_Q, 3, d_temp, 1, &beta, d_proj, 1));

    // Copy projection back
    std::vector<float> h_proj(3);
    CUDA_CHECK(hipMemcpy(h_proj.data(), d_proj, 3*sizeof(float), hipMemcpyDeviceToHost));

    std::cout << "Projection of b onto subspace spanned by v1 and v2:\n";
    std::cout << "(" << h_proj[0] << ", " << h_proj[1] << ", " << h_proj[2] << ")\n";

    hipFree(d_Q); hipFree(d_b); hipFree(d_proj); hipFree(d_temp);
    hipblasDestroy(handle);

    return 0;
}

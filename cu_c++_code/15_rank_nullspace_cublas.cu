#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define CUDA_CHECK(x) if((x)!=hipSuccess){ \
    std::cerr<<"CUDA error at "<<__LINE__<<": "<<hipGetErrorString(x)<<std::endl; return -1;}
#define CUBLAS_CHECK(x) if((x)!=HIPBLAS_STATUS_SUCCESS){ \
    std::cerr<<"cuBLAS error at "<<__LINE__<<std::endl; return -1;}

int main() {
    // Matrix dimensions
    int m = 2, n = 3;

    // Column-major matrix A
    std::vector<float> h_A = {
        1,4,   // col 0
        2,5,   // col 1
        3,6    // col 2
    };

    // Device memory
    float *d_A, *d_AtA;
    CUDA_CHECK(hipMalloc(&d_A, m*n*sizeof(float)));
    CUDA_CHECK(hipMemcpy(d_A, h_A.data(), m*n*sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMalloc(&d_AtA, n*n*sizeof(float)));

    // cuBLAS handle
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    float alpha = 1.0f, beta = 0.0f;

    // Compute AtA = A^T * A
    // A: m x n, AtA: n x n
    CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                             n, n, m, &alpha,
                             d_A, m,
                             d_A, m,
                             &beta, d_AtA, n));

    // Copy AtA to host
    std::vector<float> h_AtA(n*n);
    CUDA_CHECK(hipMemcpy(h_AtA.data(), d_AtA, n*n*sizeof(float), hipMemcpyDeviceToHost));

    std::cout << "A^T * A = \n";
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++) std::cout << h_AtA[i + j*n] << " "; // column-major
        std::cout << "\n";
    }

    // Approximate rank by counting non-zero diagonals
    float eps = 1e-6;
    int rank = 0;
    for(int i=0;i<n;i++) if(fabs(h_AtA[i + i*n]) > eps) rank++;
    std::cout << "Approximate Rank of A = " << rank << std::endl;

    // Cleanup
    hipFree(d_A); hipFree(d_AtA);
    hipblasDestroy(handle);

    return 0;
}

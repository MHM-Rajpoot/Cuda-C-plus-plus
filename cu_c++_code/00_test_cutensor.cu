#include <hiptensor.h>
#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    hiptensorHandle_t handle;
    hiptensorStatus_t status = hiptensorCreate(&handle);

    if (status == HIPTENSOR_STATUS_SUCCESS) {
        std::cout << "✅ cuTENSOR initialized successfully!" << std::endl;
    } else {
        std::cout << "❌ cuTENSOR init failed: " << status << std::endl;
    }

    hiptensorDestroy(handle); // cleanup
    return 0;
}

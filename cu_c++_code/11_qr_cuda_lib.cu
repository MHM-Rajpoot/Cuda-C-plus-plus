#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

#define CUDA_CHECK(x) if((x)!=hipSuccess){ \
    std::cerr<<"CUDA error at "<<__LINE__<<": "<<hipGetErrorString(x)<<std::endl; return -1;}
#define CUSOLVER_CHECK(x) if((x)!=HIPSOLVER_STATUS_SUCCESS){ \
    std::cerr<<"cuSOLVER error at "<<__LINE__<<std::endl; return -1;}

int main() {
    int m=3, n=2; // matrix size
    std::vector<float> h_A = {12, -51,
                               6, 167,
                              -4, 24}; // row-major

    float *d_A; CUDA_CHECK(hipMalloc(&d_A, m*n*sizeof(float)));
    CUDA_CHECK(hipMemcpy(d_A, h_A.data(), m*n*sizeof(float), hipMemcpyHostToDevice));

    hipsolverHandle_t solverH;
    CUSOLVER_CHECK(hipsolverDnCreate(&solverH));

    int lwork=0;
    CUSOLVER_CHECK(hipsolverDnSgeqrf_bufferSize(solverH, m, n, d_A, m, &lwork));

    float *d_tau, *d_work; int *devInfo;
    CUDA_CHECK(hipMalloc(&d_tau, n*sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_work, lwork*sizeof(float)));
    CUDA_CHECK(hipMalloc(&devInfo, sizeof(int)));

    // Compute QR factorization
    CUSOLVER_CHECK(hipsolverDnSgeqrf(solverH, m, n, d_A, m, d_tau, d_work, lwork, devInfo));

    // Copy back results
    std::vector<float> h_R(m*n);
    std::vector<float> h_tau(n);
    CUDA_CHECK(hipMemcpy(h_R.data(), d_A, m*n*sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_tau.data(), d_tau, n*sizeof(float), hipMemcpyDeviceToHost));

    std::cout << "R (upper-triangular part of A):\n";
    for(int i=0;i<m;i++){
        for(int j=0;j<n;j++) std::cout<<h_R[i + j*m]<<" ";
        std::cout<<std::endl;
    }

    hipFree(d_A); hipFree(d_tau); hipFree(d_work); hipFree(devInfo);
    hipsolverDnDestroy(solverH);
    return 0;
}
